#include "hip/hip_runtime.h"
// module load CUDA/11.1.1-GCC-10.2.0

#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"

#define COLOR_CHANNELS 0
#define BLOCK_SIZE 256

__device__ void to_YUV_color_space(unsigned char *image_in, int width, int height)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    while (tid < width * height) {
        int idx = tid * 3;
        
        unsigned char r = image_in[idx];
        unsigned char g = image_in[idx + 1];
        unsigned char b = image_in[idx + 2];
        
        float y = 0.299f * r + 0.587f * g + 0.114f * b;
        float u = -0.168736f *r + -0.331264f * g + 0.5f * b + 128.0f;
        float v = 0.5f * r + -0.418688f * g + -0.081312f * b + 128.0f;
        
        image_in[idx] = (unsigned char)fminf(fmaxf(y, 0.0f), 255.0f);
        image_in[idx + 1] = (unsigned char)fminf(fmaxf(u, 0.0f), 255.0f);
        image_in[idx + 2] = (unsigned char)fminf(fmaxf(v, 0.0f), 255.0f);
        
        tid += gridDim.x * blockDim.x;
    }
}

__device__ void luminance_histogram(unsigned char *image_in, int width, int height, int *histogram)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    while (tid < width * height)
    {
        int idx = tid * 3;
        unsigned char luminance = (int)image_in[idx];
        atomicAdd(&histogram[luminance], 1);
        tid += gridDim.x * blockDim.x;
    }
}

__global__ void histogram_normalization(unsigned char *image_in, int width, int height, int *histogram)
{
    to_YUV_color_space(image_in, width, height);
    luminance_histogram(image_in, width, height, histogram);
}

__global__ void cumulative_histogram(int* histogram, int len)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid == 0)
    {
        for (int i = 1; i < len; i++)
        {
            histogram[i] += histogram[i - 1];
        }
    }
}

__global__ void new_luminance(unsigned char* luminance, int *c_hist, int width, int height)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int min_cdf;
    if (tid == 0)
    {
        min_cdf = 0;
        for (int i = 0; i < 256; i++)
        {
            if (c_hist[i] > 0)
            {
                min_cdf = c_hist[i];
                break;
            }
        }
    }
    __syncthreads();
    if (tid < 256)
    {
        float scale = 255.0f / (width * height - min_cdf);
        luminance[tid] = (c_hist[tid] - min_cdf) * scale;
    }
}

__device__ void set_new_luminances(unsigned char *image_in, int width, int height, unsigned char *luminances)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    while (tid < width * height)
    {
        int idx = tid * 3;
        int l_value = (int) image_in[idx];
        image_in[idx] = luminances[l_value];
        tid += gridDim.x * blockDim.x;
    }
}

__device__ void to_RGB_color_space(unsigned char *image_in, int width, int height)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    while (tid < width * height)
    {
        int idx = tid * 3;
        double y = image_in[idx];
        double u = image_in[idx + 1] - 128; 
        double v = image_in[idx + 2] - 128;

        double r = y + v * 1.402;
        double g = y - u * 0.344136 - v * 0.714136;
        double b = y + u * 1.772;

        image_in[idx] = (unsigned char)fminf(fmaxf(r, 0.0f), 255.0f);
        image_in[idx + 1] = (unsigned char)fminf(fmaxf(g, 0.0f), 255.0f);
        image_in[idx + 2] = (unsigned char)fminf(fmaxf(b, 0.0f), 255.0f);

        tid += gridDim.x * blockDim.x;
    }
}

__global__ void generate_final_image(unsigned char *in_image, int width, int height, unsigned char *luminances)
{
    set_new_luminances(in_image, width, height, luminances);
    to_RGB_color_space(in_image, width, height);
}

void print_image(unsigned char *arr, int len)
{
    for (int i = 0; i < len; i+=3)
    {
        printf("(%d, %d, %d), ", arr[i], arr[i + 1], arr[i + 2]);
    }
    printf("\n");
}

void print_array(int *arr, int len)
{
    for (int i = 0; i < len; i++)
    {
        printf("%d, ", arr[i]);
    }
    printf("\n");
}

void print_array(unsigned char *arr, int len)
{
    for (int i = 0; i < len; i++)
    {
        printf("%d, ", arr[i]);
    }
    printf("\n");
}

int main(int argc, char *argv[])
{

    char *image_in_name = argv[1];
    int width, height, cpp;
    unsigned char *host_image = stbi_load(image_in_name, &width, &height, &cpp, COLOR_CHANNELS);
    int image_size = width * height;

    unsigned char *device_image;
    checkCudaErrors(hipMalloc((void **)&device_image, image_size * 3 * sizeof(unsigned char)));

    // HISTOGRAM
    int *host_histogram = (int *)calloc(256, sizeof(int));
    int *device_histogram;
    checkCudaErrors(hipMalloc((void **)&device_histogram, 256 * sizeof(int)));

    // LUMINANCE
    unsigned char *host_luminance = (unsigned char *)calloc(256, sizeof(unsigned char));
    unsigned char *device_luminance;
    checkCudaErrors(hipMalloc((void **)&device_luminance, 256 * sizeof(unsigned char)));

    checkCudaErrors(hipMemcpy(device_image, host_image, image_size * 3 * sizeof(unsigned char), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(device_histogram, host_histogram, 256 * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(device_luminance, host_luminance, 256 * sizeof(unsigned char), hipMemcpyHostToDevice));

    dim3 blockSize(BLOCK_SIZE);
    dim3 gridSize((image_size + blockSize.x - 1)/blockSize.x);

    // Measure compute time
    float elapsedTime;
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start));

    histogram_normalization<<<gridSize, blockSize>>>(device_image, width, height, device_histogram);
    cumulative_histogram<<<1, 32>>>(device_histogram, 256);
    new_luminance<<<1, 256>>>(device_luminance, device_histogram, width, height);
    generate_final_image<<<gridSize, blockSize>>>(device_image, width, height, device_luminance);
    checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));

    // Copy solution back to host
    checkCudaErrors(hipMemcpy(host_image, device_image, image_size * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(host_histogram, device_histogram, 256 * sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(host_luminance, device_luminance, 256 * sizeof(unsigned char), hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(device_image));
    checkCudaErrors(hipFree(device_histogram));
    checkCudaErrors(hipFree(device_luminance));

    if (!stbi_write_png("basic_parallel.png", width, height, 3, host_image, width * 3)) {
        printf("Failed to save image %s\n", "pasic_parallel.png");
        stbi_image_free(host_image);
        return 1;
    }

    printf("Saved modified image as %s\n", "basic_parallel.png");
    printf("Performed histogram normalization on image in %.2f(ms)", elapsedTime);

    printf("\n");
    free(host_image);
    free(host_histogram);
    free(host_luminance);

    return 0;
}
