#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

#define CUBE_SIZE 16
#define COLOR_CHANNELS 3

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"

__global__ void initialize_grids(float *u_array, float *v_array, int dim, float u_value, float v_value)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx == 0)
    {
        int center_start = dim / 2 - dim / 8;
        int center_end = dim / 2 + dim / 8;
        for (int i = 0; i < dim; ++i) {
            for (int j = 0; j < dim; ++j) {
                int idx = i * dim + j;
                if (i >= center_start && i < center_end && j >= center_start && j < center_end) {
                    u_array[idx] = u_value;
                    v_array[idx] = v_value;
                } else {
                    u_array[idx] = 1.0f;
                    v_array[idx] = 0.0f;
                }
            }
        }
    }
}

__global__ void calculate_next_step(float *u_array, float *v_array, float *new_u_array, float *new_v_array, int dim, float delta_time, float f, float k, float d_u, float d_v)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int column = blockIdx.x * blockDim.x + threadIdx.x;
    
    int r = row;
    while (r < dim)
    {
        int c = column;
        while (c < dim)
        {
            int idx = r * dim + c;
            int up_idx = ((r - 1 + dim) % dim) * dim + c;
            int down_idx = ((r + 1) % dim) * dim + c;
            int left_idx = r * dim + ((c - 1 + dim) % dim);
            int right_idx = r * dim + ((c + 1) % dim);

            float u_grad = u_array[down_idx] + u_array[up_idx] + u_array[left_idx] + u_array[right_idx] - 4.0f * u_array[idx];
            float v_grad = v_array[down_idx] + v_array[up_idx] + v_array[left_idx] + v_array[right_idx] - 4.0f * v_array[idx];
            new_u_array[idx] = u_array[idx] + delta_time * (-u_array[idx] * (v_array[idx] * v_array[idx]) + f * (1.0f - u_array[idx]) + d_u * u_grad);
            new_v_array[idx] = v_array[idx] + delta_time * (u_array[idx] * (v_array[idx] * v_array[idx]) - (f + k) * v_array[idx] + d_v * v_grad);
    
            
            c += gridDim.x * blockDim.x;
        }
        r += gridDim.y * blockDim.y;
    }
}

void render_to_image(unsigned char *image, float *V, int dim) {
    float v_min = V[0];
    float v_max = V[0];

    // First pass: find min and max
    for (int i = 1; i < dim * dim; ++i) {
        if (V[i] < v_min) v_min = V[i];
        if (V[i] > v_max) v_max = V[i];
    }

    float range = v_max - v_min;
    if (range == 0) range = 1.0f; // prevent division by zero

    // Second pass: normalize and convert to grayscale
    for (int i = 0; i < dim * dim; ++i) {
        float v_normalized = (V[i] - v_min) / range;
        unsigned char value = (unsigned char)(v_normalized * 255.0f);
        image[i * COLOR_CHANNELS + 0] = value;
        image[i * COLOR_CHANNELS + 1] = value;
        image[i * COLOR_CHANNELS + 2] = value;
    }
}

void float_print(float *array, int n)
{
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            printf("%.2f, ", array[i * n + j]);
        }
        printf("\n");
    }    
}

void char_print(unsigned char *array, int n)
{
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            printf("%d, ", array[i * n + j]);
        }
        printf("\n");
    }    
}

int main(int argc, char *argv[])
{
    int n = 1024;
    int steps = 5000;
    float delta_time = 1.0;
    float d_u = 0.16f;
    float d_v = 0.08f;
    float f = 0.06;
    float k = 0.062f;


    float *host_u_array = (float*) malloc(n * n * sizeof(float));
    float *host_v_array = (float*) malloc(n * n * sizeof(float));
    float *host_new_u_array = (float*) malloc(n * n * sizeof(float));
    float *host_new_v_array = (float*) malloc(n * n * sizeof(float));

    float *device_u_array, *device_v_array, *device_new_u_array, *device_new_v_array;

    checkCudaErrors(hipMalloc((void **)&device_u_array, n * n * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&device_v_array, n * n * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&device_new_u_array, n * n * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&device_new_v_array, n * n * sizeof(float)));

    dim3 blockSize(CUBE_SIZE, CUBE_SIZE);
    dim3 gridSize((n + CUBE_SIZE - 1) / CUBE_SIZE, (n + CUBE_SIZE - 1) / CUBE_SIZE);
    initialize_grids<<<1, 32>>>(device_u_array, device_v_array, n, 0.75f, 0.25f);
    checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipMemcpy(host_u_array, device_u_array, n * n * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(host_v_array, device_v_array, n * n * sizeof(float), hipMemcpyDeviceToHost));

    // printf("U:\n");
    // float_print(host_u_array, n);
    // printf("V:\n");
    // float_print(host_v_array, n);
    unsigned char *init_image = (unsigned char*) malloc(3 * n * n * sizeof(unsigned char));

    render_to_image(init_image, host_u_array, n);

    if (!stbi_write_png("InitImage.png", n, n, 3, init_image, n * 3)) {
        printf("Failed to save image %s\n", "InitImage.png");
        stbi_image_free(init_image);
        return 1;
    }


    for (int i = 0; i < steps; i++)
    {
        calculate_next_step<<<gridSize, blockSize>>>(device_u_array, device_v_array, device_new_u_array, device_new_v_array, n, delta_time, f, k, d_u, d_v);
        checkCudaErrors(hipGetLastError());
        float *temp_pointer = device_u_array;
        device_u_array = device_new_u_array;
        device_new_u_array = temp_pointer;
        temp_pointer = device_v_array;
        device_v_array = device_new_v_array;
        device_new_v_array = temp_pointer;
    }

    checkCudaErrors(hipMemcpy(host_u_array, device_u_array, n * n * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(host_v_array, device_v_array, n * n * sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(device_v_array));
    checkCudaErrors(hipFree(device_u_array));
    checkCudaErrors(hipFree(device_new_u_array));
    checkCudaErrors(hipFree(device_new_v_array));

    // printf("RESULT U:\n");
    // float_print(host_u_array, n);
    // printf("RESULT V:\n");
    // float_print(host_v_array, n);

    unsigned char *result_image = (unsigned char *)malloc(COLOR_CHANNELS * n * n * sizeof(unsigned char));
    render_to_image(result_image, host_v_array, n);

    free(host_u_array);
    free(host_v_array);
    free(host_new_u_array);
    free(host_new_v_array);

    if (!stbi_write_png("Result.png", n, n, 3, result_image, n * 3)) {
        printf("Failed to save image %s\n", "Result.png");
        stbi_image_free(result_image);
        return 1;
    }

    return 0;
}